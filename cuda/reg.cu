#include "tipl/tipl.hpp"
#include "tipl/cuda/mem.hpp"
#include "tipl/cuda/basic_image.hpp"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

float two_way_linear_cuda(const tipl::image<3,float>& I,
                          const tipl::vector<3>& Ivs,
                         const tipl::image<3,float>& J,
                         const tipl::vector<3>& Jvs,
                         tipl::transformation_matrix<float>& T,
                         tipl::reg::reg_type reg_type,
                         bool& terminated,
                         tipl::affine_transform<float>* arg_min,
                         const float* bound)
{    
    float result(0);
    std::cout << "linear registration using GPU" << std::endl;
    try{
        result = tipl::reg::two_way_linear_mr<tipl::reg::mutual_information_cuda>
                (I,Ivs,J,Jvs,T,reg_type,terminated,arg_min,bound);
    }
    catch(std::runtime_error& er)
    {
        std::cout << "ERROR: " << er.what() << std::endl;
        std::cout << "switch to CPU" << std::endl;
        result = tipl::reg::two_way_linear_mr<tipl::reg::mutual_information>
                    (I,Ivs,J,Jvs,T,reg_type,terminated,arg_min,bound);
    }
    std::cout << "T:" << T;
    return result;
}


void cdm2_cuda(const tipl::image<3>& It,
               const tipl::image<3>& It2,
               const tipl::image<3>& Is,
               const tipl::image<3>& Is2,
               tipl::image<3,tipl::vector<3> >& d,
               bool& terminated,
               tipl::reg::cdm_param param)
{
    std::cout << "normalization using GPU" << std::endl;
    tipl::device_image<3> dIt(It),dIt2(It2),dIs(Is),dIs2(Is2);
    tipl::device_image<3,tipl::vector<3> > dd(It.shape());
    tipl::reg::cdm2_cuda(dIt,dIt2,dIs,dIs2,dd,terminated,param);

    try{
        tipl::reg::cdm2_cuda(dIt,dIt2,dIs,dIs2,dd,terminated,param);
    }
    catch(std::runtime_error& er)
    {
        std::cout << "ERROR: " << er.what() << std::endl;
        std::cout << "switch to CPU" << std::endl;
        tipl::reg::cdm2(It,It2,Is,Is2,d,terminated,param);
        return;
    }
    d.resize(It.shape());
    dd.vector().copy_to(d);
}

float linear_mr(tipl::const_pointer_image<3,float> I,
                         const tipl::vector<3>& Ivs,
                         tipl::const_pointer_image<3,float> J,
                         const tipl::vector<3>& Jvs,
                         tipl::affine_transform<float>& T,
                         tipl::reg::reg_type reg_type,
                         bool& terminated,
                         double precision,
                         const float* bound)
{
    std::cout << "linear registration using GPU" << std::endl;
    return tipl::reg::linear_mr<tipl::reg::mutual_information_cuda>(I,Ivs,J,Jvs,T,reg_type,terminated,precision,bound);
}

float linear_mr_uint8(tipl::const_pointer_image<3,unsigned char> I,
                         const tipl::vector<3>& Ivs,
                         tipl::const_pointer_image<3,unsigned char> J,
                         const tipl::vector<3>& Jvs,
                         tipl::affine_transform<float>& T,
                         tipl::reg::reg_type reg_type,
                         bool& terminated,
                         double precision,
                         const float* bound)
{
    std::cout << "linear registration using GPU" << std::endl;
    return tipl::reg::linear_mr<tipl::reg::mutual_information_cuda>(I,Ivs,J,Jvs,T,reg_type,terminated,precision,bound);
}
